#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <math.h>

#define WIDTH 2560
#define HEIGHT 1440
#define MAX_ITER 65535
#define COLOR_DEPTH 65535  

// Global variables for position & zoom (updated via Zig)
__device__ float offset_x = -0.5f;
__device__ float offset_y = 0.0f;
__device__ float zoom = 1.0f;

__device__ uint16_t mandelbrot(float x, float y) {
    float zx = 0.0, zy = 0.0;
    uint16_t iter = 0;

    while (zx * zx + zy * zy < 4.0f && iter < MAX_ITER) {
        float temp = zx * zx - zy * zy + x;
        zy = 2.0f * zx * zy + y;
        zx = temp;
        iter++;
    }
    return iter;
}

extern "C" __global__ void compute_mandelbrot(uint16_t *pixels, float new_offset_x, float new_offset_y, float new_zoom) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px >= WIDTH || py >= HEIGHT) return;

    // Update global offsets dynamically
    offset_x = new_offset_x;
    offset_y = new_offset_y;
    zoom = new_zoom;

    // Compute scaled coordinates based on offset & zoom
    float x0 = (px - WIDTH / 2.0f) / (WIDTH / 2.0f) * (3.5f / zoom) + offset_x;
    float y0 = (py - HEIGHT / 2.0f) / (HEIGHT / 2.0f) * (2.0f / zoom) + offset_y;

    uint16_t iter = mandelbrot(x0, y0);

    // Nonlinear color mapping
    float t = logf((float)iter + 1) / logf((float)MAX_ITER);
    uint16_t r = (uint16_t)(t * COLOR_DEPTH);
    uint16_t g = (uint16_t)((t * 0.8f) * COLOR_DEPTH);
    uint16_t b = (uint16_t)((t * 0.5f) * COLOR_DEPTH);

    int index = (py * WIDTH + px) * 3;
    pixels[index] = r;
    pixels[index + 1] = g;
    pixels[index + 2] = b;
}

extern "C" void launch_mandelbrot(uint16_t *pixels, float offset_x, float offset_y, float zoom) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    compute_mandelbrot<<<numBlocks, threadsPerBlock>>>(pixels, offset_x, offset_y, zoom);
    hipDeviceSynchronize();
}
