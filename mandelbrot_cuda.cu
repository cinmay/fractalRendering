#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <math.h>

#define WIDTH 2560
#define HEIGHT 1440
#define MAX_ITER 65535
#define COLOR_DEPTH 65535  

// Global variables for position & zoom (updated via Zig)
__device__ float offset_x = -0.5f;
__device__ float offset_y = 0.0f;
__device__ float zoom = 1.0f;

__device__ uint16_t mandelbrot(float x, float y) {
    float zx = 0.0, zy = 0.0;
    uint16_t iter = 0;

    while (zx * zx + zy * zy < 4.0f && iter < MAX_ITER) {
        float temp = zx * zx - zy * zy + x;
        zy = 2.0f * zx * zy + y;
        zx = temp;
        iter++;
    }
    return iter;
}

extern "C" __global__ void compute_mandelbrot(uint16_t *pixels, float new_offset_x, float new_offset_y, float new_zoom) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px >= WIDTH || py >= HEIGHT) return;

    // Update global offsets dynamically
    offset_x = new_offset_x;
    offset_y = new_offset_y;
    zoom = new_zoom;

    // Compute scaled coordinates based on offset & zoom
    float x0 = (px - WIDTH / 2.0f) / (WIDTH / 2.0f) * (3.5f / zoom) + offset_x;
    float y0 = (py - HEIGHT / 2.0f) / (HEIGHT / 2.0f) * (2.0f / zoom) + offset_y;

    uint16_t iter = mandelbrot(x0, y0);

    // Compute a normalized value t based on iterations
    float t = logf((float)iter + 1.0f) / logf((float)MAX_ITER);

    float r, g, b;
    if (t < 0.5f) {
        // Interpolate from deep blue (0,0,139) to neon pink (255,20,147)
        float frac = t / 0.5f;
        r = (1.0f - frac) * 0.0f + frac * 255.0f;
        g = (1.0f - frac) * 0.0f + frac * 20.0f;
        b = (1.0f - frac) * 139.0f + frac * 147.0f;
    } else {
        // Interpolate from neon pink (255,20,147) to neon orange (255,165,0)
        float frac = (t - 0.5f) / 0.5f;
        r = (1.0f - frac) * 255.0f + frac * 255.0f; // remains 255
        g = (1.0f - frac) * 20.0f + frac * 165.0f;
        b = (1.0f - frac) * 147.0f + frac * 0.0f;
    }

    // Scale colors from 0-255 to 0-COLOR_DEPTH (65535)
    uint16_t r_scaled = (uint16_t)(r * 257.0f);
    uint16_t g_scaled = (uint16_t)(g * 257.0f);
    uint16_t b_scaled = (uint16_t)(b * 257.0f);

    int index = (py * WIDTH + px) * 3;
    pixels[index] = r_scaled;
    pixels[index + 1] = g_scaled;
    pixels[index + 2] = b_scaled;
}

extern "C" void launch_mandelbrot(uint16_t *pixels, float offset_x, float offset_y, float zoom) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    compute_mandelbrot<<<numBlocks, threadsPerBlock>>>(pixels, offset_x, offset_y, zoom);
    hipDeviceSynchronize();
}
